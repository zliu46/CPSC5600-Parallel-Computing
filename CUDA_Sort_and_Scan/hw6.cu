/**
* @file hw6.cu
 * @brief CUDA Bitonic Sort and Parallel Prefix Scan for CSV Data
 *
 * This program sorts `x` values using **Bitonic Sort** and computes the cumulative sum
 * of `y` values using a **Parallel Prefix Scan**. The input is a CSV file with (x, y) pairs,
 * and the output is a sorted CSV with cumulative y-values.
 *
 * @Author: Zhou Liu - Seattle University, CPSC 5600, Winter 2025
 */
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <string>
using namespace std;

// Maximum threads per block
const int MAX_BLOCK_SIZE = 1024;

// Struct to hold (x, y) values along with cumulative Y values and original row index
struct X_Y {
    float x, y; // x and y values from CSV file
    float cumulativeY; // Cumulative sum of y values
    size_t originalRow; // Original row index in the CSV file
};

/**
 * @brief CUDA Kernel for Bitonic Sort.
 * @param data Device array of X_Y structures.
 * @param k Outer loop iteration for bitonic sort.
 * @param j Inner loop iteration for bitonic sort.
 * @param size Number of elements in the data array.
 */
__global__ void bitonic(X_Y *data, int k, int j, int size) {
    // Compute global thread index
	int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Compute partner index for comparison
	int ixj = i ^ j;

    // Prevent out-of-bounds memory access
	if(ixj >= size){
		return;
	}

    // Perform Bitonic Sorting comparison and swap
	if (ixj > i) {
	    // Sorting in ascending order for even-indexed blocks
		if ((i & k) == 0 && data[i].x > data[ixj].x)
		{
			X_Y temp = data[i];
			data[i] = data[ixj];
			data[ixj] = temp;
		}
	    // Sorting in descending order for odd-indexed blocks
		if ((i & k) != 0 && data[i].x < data[ixj].x)
		{
			X_Y temp = data[i];
			data[i] = data[ixj];
			data[ixj] = temp;
		}
	}
}

/**
 * @brief CUDA Kernel for Parallel Prefix Scan (Cumulative Sum).
 * @param data Device array of X_Y structures.
 * @param size Number of elements in the data array.
 */
__global__ void scan(X_Y *data, int size, int tier) {
    __shared__ X_Y local[MAX_BLOCK_SIZE];
    if(tier == 1){
        int gindex = blockDim.x * blockIdx.x + threadIdx.x;
        if(gindex >= size){
            return;
        }
        int index = threadIdx.x;
        local[index] = data[gindex];
        local[index].cumulativeY = local[index].y;
        // Inclusive scan within the block
        for (int stride = 1; stride < blockDim.x; stride *= 2) {
            __syncthreads();
            float addend = 0;
            if (stride <= index)
                addend = local[index - stride].cumulativeY;
            __syncthreads();
            local[index].cumulativeY += addend;
        }
        data[gindex] = local[index];
    }
    if(tier == 2){
        int gindex = ((blockDim.x  * (blockDim.x * blockIdx.x + threadIdx.x)) + blockDim.x) - 1;
        if(gindex >= size){
            return;
        }
        int index = threadIdx.x;
        local[index] = data[gindex];
        for (int stride = 1; stride < blockDim.x; stride *= 2) {
            __syncthreads();
            float addend = 0;
            if (stride <= index)
                addend = local[index - stride].cumulativeY;
            __syncthreads();
            local[index].cumulativeY += addend;
        }
        __syncthreads();
        data[gindex] = local[index];
    }
}

/**
 * @brief Kernel for block-level cleanup in scan (propagates sums across blocks).
 * @param data Device array of X_Y structures.
 * @param blockSums Prefix sum of each block.
 * @param size Number of elements in the data array.
 */
__global__ void clean(X_Y *data, int size){
    int gindex = blockDim.x * blockIdx.x + threadIdx.x;
    int lastIndex = blockDim.x * blockIdx.x - 1;
    if(blockIdx.x != 0 && threadIdx.x != (blockDim.x-1)){ // Add previous block's sum
        data[gindex].cumulativeY += data[lastIndex].cumulativeY;
    }
}

/**
 * @brief Reads a CSV file and loads data into a vector.
 * @param filename Name of the CSV file.
 * @param data Vector to store parsed X_Y structures.
 */
void readCSV(const string &filename, vector<X_Y> &data) {
    ifstream file(filename);
    if (!file.is_open()) {
        cerr << "Error opening file: " << filename << endl;
        exit(EXIT_FAILURE);
    }

    string line;
	getline(file, line);
    size_t index = 1;
    while (getline(file, line)) {
        istringstream iss(line);
        X_Y point;
        char comma;  // to store the comma between x and y values
        iss >> point.x >> comma >> point.y;
        point.originalRow = index;
        data.push_back(point);
        index++;
    }

    file.close();
}

/**
 * @brief Writes the processed data to an output CSV file.
 * @param filename Output file name.
 * @param data Array of X_Y structures.
 * @param size Number of elements in the data array.
 */
void writeCSV(const string &filename, const X_Y* data, size_t size) {
    ofstream file(filename);
    if (!file.is_open()) {
        cerr << "Error opening file: " << filename << endl;
        exit(EXIT_FAILURE);
    }

    file << "x value, y value, cumulative Y value, original row number\n";

    for (size_t i = 0; i < size; i++)
    {
        file << data[i].x << "," << data[i].y << "," << data[i].cumulativeY << "," << data[i].originalRow << endl;
    }

    file.close();
}

/**
 * @brief Error handling function for CUDA API calls.
 * @param status CUDA error code
 */
void auto_throw(hipError_t status) {
    if (status != hipSuccess) {
        string message = "ERROR: '";
        message += hipGetErrorString(status);
        message += "'\n";
        throw runtime_error(message);
    }
}

/**
 * @brief Main function for CUDA-based Bitonic Sort and Prefix Scan.
 */
int main() {
    //Data vector to hold CSV data file
    vector<X_Y> vectorData;
    string filename = "x_y.csv";
    string outFileName = "x_y_scan.csv";
    const int tier = 2;
    readCSV(filename, vectorData);
	X_Y *data; //Holds the CSV data file on GPU

    //Allocate memory on GPU
	auto_throw(hipMallocManaged(&data, vectorData.size() * sizeof(X_Y)));

    //Divide the number of blocks based on vector data size
    const int numOfBlocks = (vectorData.size() + MAX_BLOCK_SIZE - 1) / MAX_BLOCK_SIZE;

    // Copy data from vector to CUDA Unified Memory
	for(int i = 0; i < vectorData.size(); i++){
		data[i] = vectorData[i];
	}

    // Perform Bitonic Sort
    for (int k = 2; k <= vectorData.size(); k *= 2) {
        for (int j = k / 2; j > 0; j /= 2) {
            bitonic<<<numOfBlocks, MAX_BLOCK_SIZE>>>(data, k, j, vectorData.size());
            //Synchronize the threads over blocks
            auto_throw(hipDeviceSynchronize());
        }
    }

    // Perform Prefix Scan
    for(int i = 1; i < tier + 1; i++ ){
        scan<<<numOfBlocks, MAX_BLOCK_SIZE>>>(data, vectorData.size(), i);
        auto_throw(hipDeviceSynchronize());
    }
    // Perform Cleanup
    clean<<<numOfBlocks, MAX_BLOCK_SIZE>>>(data, vectorData.size());
    auto_throw(hipDeviceSynchronize());

    // Write results to CSV
    writeCSV(outFileName, data, vectorData.size());

    // Free GPU memory
	hipFree(data);
	return 0;
}
